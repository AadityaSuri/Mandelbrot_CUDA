#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/bitmap.h"
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <chrono>

extern "C" {
    void generateBitmapImage(unsigned char* image, int height, int width, char* imageFileName);
    void writePixel(unsigned char* image, int height, int width, int i, int j, int r, int g, int b);
}

int colorMap[16][3] = {    
    {60, 30, 15},
    {25, 7, 26},
    {9, 1, 47},
    {4, 4, 73},
    {0, 7, 100},
    {12, 44, 138},
    {24, 82, 177},
    {57, 125, 209},
    {134, 181, 229},
    {211, 236, 248},
    {241, 233, 191},
    {248, 201, 95},
    {255, 170, 0},
    {204, 128, 0},
    {153, 87, 0},
    {106, 52, 3}
};

__constant__ int d_colorMap[16][3];

__global__ void mandelbrotKernel(unsigned char* image, int height, int width, 
                                 double x_min, double x_max, double y_min, double y_max, 
                                 int max_iteration) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < height && j < width) {
        double x = 0;
        double y = 0;
        int iteration = 0;
        double x_scaled = x_min + j * (x_max - x_min) / (width - 1.0);
        double y_scaled = y_min + i * (y_max - y_min) / (height - 1.0);
        
        while (x*x + y*y <= 4 && iteration < max_iteration) {
            double xtemp = x*x - y*y + x_scaled;
            y = 2*x*y + y_scaled;
            x = xtemp;
            iteration++;
        }
        
        int r = 0, g = 0, b = 0;
        if (iteration < max_iteration) {
            int colorIndex = iteration % 16;
            r = d_colorMap[colorIndex][0];
            g = d_colorMap[colorIndex][1];
            b = d_colorMap[colorIndex][2];
        }
        
        int index = (i * width + j) * BYTES_PER_PIXEL;
        image[index + 2] = (unsigned char) r;  // Red
        image[index + 1] = (unsigned char) g;  // Green
        image[index] = (unsigned char) b;      // Blue
    }
}

void drawMandelbrot_cuda(unsigned char *image, int height, int width, double centerX, double centerY, 
                         double zoomFactor, char *imageFileName) {
    double padding = 0.025;
    double xSpan = 1.0 / zoomFactor;
    double ySpan = 1.0 / zoomFactor;
    double x_min = centerX - (xSpan / 2.0) - padding;
    double x_max = centerX + (xSpan / 2.0) + padding;
    double y_min = centerY - (ySpan / 2.0) - padding;
    double y_max = centerY + (ySpan / 2.0) + padding;
    int max_iteration = 1000;

    // Allocate device memory
    unsigned char *d_image;
    hipMalloc(&d_image, height * width * BYTES_PER_PIXEL * sizeof(unsigned char));

    // Copy colorMap to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_colorMap), colorMap, sizeof(int) * 16 * 3);

    // Set up grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
                  (height + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    mandelbrotKernel<<<gridSize, blockSize>>>(d_image, height, width, x_min, x_max, y_min, y_max, max_iteration);

    // Copy result back to host
    hipMemcpy(image, d_image, height * width * BYTES_PER_PIXEL * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_image);

    // Generate bitmap image
    generateBitmapImage((unsigned char*) image, height, width, imageFileName);
    // printf("Image generated using CUDA: %s\n", imageFileName);

    // Check for any CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}


void drawMandelbrot(unsigned char *image, int height, int width, double centerX, double centerY, double zoomFactor, char *imageFileName) {
    double padding = 0.025;

    // calculate the x and y spans
    double xSpan = 1.0 / zoomFactor;
    double ySpan = 1.0 / zoomFactor;

    // calculate min and max x and y based on the center coordinates and zoom factor
    double x_min = centerX - (xSpan / 2.0) - padding;
    double x_max = centerX + (xSpan / 2.0) + padding;
    double y_min = centerY - (ySpan / 2.0) - padding;
    double y_max = centerY + (ySpan / 2.0) + padding;

    int i, j;
    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            double x = 0;
            double y = 0;

            int iteration = 0;
            int max_iteration = 1000;

            double x_scaled = x_min + j * (x_max - x_min) / (width - 1.0);
            double y_scaled = y_min + i * (y_max - y_min) / (height - 1.0);

            while (x*x + y*y <= 4 && iteration < max_iteration) {
                double xtemp = x*x - y*y + x_scaled;
                y = 2*x*y + y_scaled;
                x = xtemp;
                iteration++;
            }

            int r = 0;
            int g = 0;
            int b = 0;

            if (iteration < max_iteration) {
                int i = iteration % 16;

                r = colorMap[i][0];
                g = colorMap[i][1];
                b = colorMap[i][2];
            }

            writePixel((unsigned char*) image, height, width, i, j, r, g, b);
        }
    }

    generateBitmapImage((unsigned char*) image, height, width, imageFileName);
    // printf("Image generated!!");
}

int main() {
    int height = 1200;
    int width = 1500;
    unsigned char image[height][width][BYTES_PER_PIXEL];

    // **********************************************************************
    // ************************ LINEAR MANDELBROT **************************
    // **********************************************************************

    auto start_cpu = std::chrono::high_resolution_clock::now();

    char* imageFileName = (char*) "output/linear/rand.bmp";
    drawMandelbrot((unsigned char*) image, height, width, -0.7, -0.375, 3.0, imageFileName);

    char* seahorseFileName = (char*) "output/linear/seahorseValley.bmp";
    drawMandelbrot((unsigned char*) image, height, width, -0.747, 0.1, 1/0.005, seahorseFileName);

    // Elephant Valley
    char* elephantFileName = (char*) "output/linear/elephantValley.bmp";
    drawMandelbrot((unsigned char*) image, height, width, 0.275, 0.0, 1/0.01, elephantFileName);

    // Triple Spiral Valley
    char* tripleSpiralFileName = (char*) "output/linear/tripleSpiralValley.bmp";
    drawMandelbrot((unsigned char*) image, height, width, -0.088, 0.654, 1/0.005, tripleSpiralFileName);

    // Mini Mandelbrot
    char* miniMandelbrotFileName = (char*) "output/linear/miniMandelbrot.bmp";
    drawMandelbrot((unsigned char*) image, height, width, -1.768, 0.001, 1/0.001, miniMandelbrotFileName);

    // full Mandelbrot
    char* fullMandelbrotFileName = (char*) "output/linear/fullMandelbrot.bmp";
    drawMandelbrot((unsigned char*) image, height, width, -0.75, 0.0, 0.35, fullMandelbrotFileName);

    auto end_cpu = std::chrono::high_resolution_clock::now();
    auto duration_cpu = std::chrono::duration_cast<std::chrono::milliseconds>(end_cpu - start_cpu).count();
    printf("CPU time: %d ms\n", duration_cpu);

    // **********************************************************************
    // ************************ CUDA MANDELBROT *****************************
    // **********************************************************************

    auto start_cuda = std::chrono::high_resolution_clock::now();

    char* imageFileName_cuda = (char*) "output/cuda/rand.bmp";
    drawMandelbrot_cuda((unsigned char*) image, height, width, -0.7, -0.375, 3.0, imageFileName_cuda);

    char* seahorseFileName_cuda = (char*) "output/cuda/seahorseValley.bmp";
    drawMandelbrot_cuda((unsigned char*) image, height, width, -0.747, 0.1, 1/0.005, seahorseFileName_cuda);

    // Elephant Valley
    char* elephantFileName_cuda = (char*) "output/cuda/elephantValley.bmp";
    drawMandelbrot_cuda((unsigned char*) image, height, width, 0.275, 0.0, 1/0.01, elephantFileName_cuda);

    // Triple Spiral Valley
    char* tripleSpiralFileName_cuda = (char*) "output/cuda/tripleSpiralValley.bmp";
    drawMandelbrot_cuda((unsigned char*) image, height, width, -0.088, 0.654, 1/0.005, tripleSpiralFileName_cuda);

    // Mini Mandelbrot
    char* miniMandelbrotFileName_cuda = (char*) "output/cuda/miniMandelbrot.bmp";
    drawMandelbrot_cuda((unsigned char*) image, height, width, -1.768, 0.001, 1/0.001, miniMandelbrotFileName_cuda);

    // full Mandelbrot
    char* fullMandelbrotFileName_cuda = (char*) "output/cuda/fullMandelbrot.bmp";
    drawMandelbrot_cuda((unsigned char*) image, height, width, -0.75, 0.0, 0.35, fullMandelbrotFileName_cuda);

    auto end_cuda = std::chrono::high_resolution_clock::now();
    auto duration_cuda = std::chrono::duration_cast<std::chrono::milliseconds>(end_cuda - start_cuda).count();
    printf("CUDA time: %d ms\n", duration_cuda);

    return 0;
}

